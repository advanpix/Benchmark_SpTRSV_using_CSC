#include "common.h"
#include "mmio.h"

#include "spts_syncfree_serialref.h"
#include "spts_syncfree_cuda.h"

int main(int argc, char ** argv)
{
    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char  *precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = (char *)"32-bit Single Precision";
    }
    else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = (char *)"64-bit Double Precision";
    }
    else
    {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }

    printf("PRECISION = %s\n", precision);
    printf("Benchmark REPEAT = %i\n", BENCH_REPEAT);
    printf("---------------------------------------------------------------------------------------------\n");

    int m, n, nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;

    //ex: ./spmv webbase-1M.mtx
    int argi = 1;

    char  *filename;
    if(argc > argi)
    {
        filename = argv[argi];
        argi++;
    }
    printf("-------------- %s --------------\n", filename);

    // read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;

    // load matrix
    if ((f = fopen(filename, "r")) == NULL)
        return -1;

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        return -2;
    }

    if ( mm_is_complex( matcode ) )
    {
        printf("Sorry, data type 'COMPLEX' is not supported.\n");
        return -3;
    }

    if ( mm_is_pattern( matcode ) )  { isPattern = 1; /*printf("type = Pattern\n");*/ }
    if ( mm_is_real ( matcode) )     { isReal = 1; /*printf("type = real\n");*/ }
    if ( mm_is_integer ( matcode ) ) { isInteger = 1; /*printf("type = integer\n");*/ }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        return -4;

    if ( mm_is_symmetric( matcode ) || mm_is_hermitian( matcode ) )
    {
        isSymmetric = 1;
        printf("input matrix is symmetric = true\n");
    }
    else
    {
        printf("input matrix is symmetric = false\n");
    }

    int *csrRowPtrA_counter = (int *)malloc((m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE *csrValA_tmp    = (VALUE_TYPE *)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++)
    {
        int idxi, idxj;
        double fval;
        int ival;
        int returnvalue;

        if (isReal)
            returnvalue = fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger)
        {
            returnvalue = fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        }
        else if (isPattern)
        {
            returnvalue = fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++)
    {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i-1];
        old_val = new_val;
    }

    nnzA = csrRowPtrA_counter[m];
    csrRowPtrA = (int *)malloc((m+1) * sizeof(int));
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m+1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    csrColIdxA = (int *)malloc(nnzA * sizeof(int));
    csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));

    if (isSymmetric)
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            }
            else
            {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    }
    else
    {
        for (int i = 0; i < nnzA_mtx_report; i++)
        {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }
/*
    // free tmp space
    free(csrColIdxA_tmp);
    free(csrValA_tmp);
    free(csrRowIdxA_tmp);
    free(csrRowPtrA_counter);

    // a small matrix
    free(csrColIdxA);
    free(csrValA);
    free(csrRowPtrA);

    m = n = 8;
    nnzA = 17;
    csrRowPtrA = (int *)malloc(sizeof(int) * (m+1));
    csrColIdxA = (int *)malloc(sizeof(int) * nnzA);
    csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));
    csrRowPtrA[0] = 0; csrRowPtrA[1] = 1; csrRowPtrA[2] = 2; csrRowPtrA[3] = 4; csrRowPtrA[4] = 6; 
    csrRowPtrA[5] = 10; csrRowPtrA[6] = 12; csrRowPtrA[7] = 15; csrRowPtrA[8] = nnzA;
    

    csrColIdxA[0] = 0;  csrColIdxA[1] = 1;  csrColIdxA[2] = 1;  csrColIdxA[3] = 2;  csrColIdxA[4] = 0;  
    csrColIdxA[5] = 3;  csrColIdxA[6] = 1;  csrColIdxA[7] = 2;  csrColIdxA[8] = 3;  csrColIdxA[9] = 4;  
    csrColIdxA[10] = 3;  csrColIdxA[11] = 5;  csrColIdxA[12] = 2;  csrColIdxA[13] = 5;  csrColIdxA[14] = 6;  
    csrColIdxA[15] = 6;  csrColIdxA[16] = 7;  
    // a small matrix stop
*/
    printf("input matrix A: ( %i, %i ) nnz = %i\n", m, n, nnzA);

    // extract L with the unit-lower triangular sparsity structure of A
    int nnzL = 0;
    int *csrRowPtrL_tmp = (int *)malloc((m+1) * sizeof(int));
    int *csrColIdxL_tmp = (int *)malloc(nnzA * sizeof(int));
    VALUE_TYPE *csrValL_tmp    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));

    int nnz_pointer = 0;
    csrRowPtrL_tmp[0] = 0;
    for (int i = 0; i < m; i++)
    {
        for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
        {
            if (csrColIdxA[j] < i)
            {
                csrColIdxL_tmp[nnz_pointer] = csrColIdxA[j];
                csrValL_tmp[nnz_pointer] = 1.0; //csrValA[j];
                nnz_pointer++;
            }
            else
            {
                break;
            }
        }

        csrColIdxL_tmp[nnz_pointer] = i;
        csrValL_tmp[nnz_pointer] = 1.0;
        nnz_pointer++;

        csrRowPtrL_tmp[i+1] = nnz_pointer;
    }

    nnzL = csrRowPtrL_tmp[m];
    printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzL);

    csrColIdxL_tmp = (int *)realloc(csrColIdxL_tmp, sizeof(int) * nnzL);
    csrValL_tmp = (VALUE_TYPE *)realloc(csrValL_tmp, sizeof(VALUE_TYPE) * nnzL);

    // run serial syncfree SpTS as a reference
    printf("---------------------------------------------------------------------------------------------\n");
    spts_syncfree_serialref(csrRowPtrL_tmp, csrColIdxL_tmp, csrValL_tmp, m, n, nnzL);

    // set device
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    printf("---------------------------------------------------------------------------------------------\n");
    printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);

    // run cuda syncfree SpTRSV
    printf("---------------------------------------------------------------------------------------------\n");
    spts_syncfree_cuda(csrRowPtrL_tmp, csrColIdxL_tmp, csrValL_tmp, m, n, nnzL);

    printf("---------------------------------------------------------------------------------------------\n");

    // done!
    free(csrColIdxA);
    free(csrValA);
    free(csrRowPtrA);

    free(csrColIdxL_tmp);
    free(csrValL_tmp);
    free(csrRowPtrL_tmp);

    return 0;
}
